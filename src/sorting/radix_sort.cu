#include "hip/hip_runtime.h"
    /*
     *
     * radix_sort.cu
     *
     */
#include <assert.h>     
 #include <time.h>
#include <time.h>
 #include <stdio.h>
 #include <stdlib.h>
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

#ifndef MAX_THREADS
#define 	MAX_THREADS 	128
#endif

#ifndef N
#define 	N 	128
#endif 


 int* r_values;
 int* d_values;
 int* t_values;

 int* d_split;
 int* d_e;
 int* d_f;
 int* d_t;

 // convert to binary string
 char* Itoa(int value, char* str, int radix) {
        static char dig[] =
                "0123456789"
                "abcdefghijklmnopqrstuvwxyz";
        int n = 0, neg = 0;
        unsigned int v;
        char* p, *q;
        char c;

        if (radix == 10 && value < 0) {
                value = -value;
                neg = 1;
        }
        v = value;
        do {
                str[n++] = dig[v%radix];
                v /= radix;
        } while (v);
        if (neg)
                str[n++] = '-';
        str[n] = '\0';

        for (p = str, q = p + (n-1); p < q; ++p, --q)
        c = *p, *p = *q, *q = c;
        return str;
}

 // initialize data set
 void Init(int* values, int i) {
        srand( time(NULL) );
        printf("\n------------------------------\n");

        if (i == 0) {
        // Uniform distribution
                printf("Data set distribution: Uniform\n");
                for (int x = 0; x < N; ++x) {
                        values[x] = rand() % 100;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 1) {
        // Gaussian distribution
        #define MEAN    100
        #define STD_DEV 5
                printf("Data set distribution: Gaussian\n");
                float r;
                for (int x = 0; x < N; ++x) {
                        r  = (rand()%3 - 1) + (rand()%3 - 1) + (rand()%3 - 1);
                        values[x] = int( round(r * STD_DEV + MEAN) );
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 2) {
        // Bucket distribution
                printf("Data set distribution: Bucket\n");
                int j = 0;
                for (int x = 0; x < N; ++x, ++j) {
                        if (j / 20 < 1)
                                values[x] = rand() % 20;
                        else if (j / 20 < 2)
                                values[x] = rand() % 20 + 20;
                        else if (j / 20 < 3)
                                values[x] = rand() % 20 + 40;
                        else if (j / 20 < 4)
                                values[x] = rand() % 20 + 60;
                        else if (j / 20 < 5)
                                values[x] = rand() % 20 + 80;
                        if (j == 100)
                                j = 0;
                        //printf("%d ", values[x]);
                }
        }
        else if (i == 3) {
        // Sorted distribution
                printf("Data set distribution: Sorted\n");
                /*for (int x = 0; x < N; ++x)
                        printf("%d ", values[x]);
                */
        }
    else if (i == 4) {
        	// Zero distribution
                printf("Data set distribution: Zero\n");
            int r = rand() % 100;
         	for (int x = 0; x < N; ++x) {
                       	values[x] = r;
                       	//printf("%d ", values[x]);
            }
        }

    // Convert to binary
        char binary_string[8];
    for (int x = 0; x < N; ++x) {
        	Itoa(r_values[x], binary_string, 2);		// convert to binary string
                r_values[x] = atoi(binary_string);		// convert to binary int
	    //printf(" %d\n", r_values[x]);
    }

       	printf("\n");
}

 // Kernel function
 __global__ static void Radix_sort(int* values, int* temp, int loop, int* split, int* e, int* f, int* t) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int remainder[N], quotient[N];
    int f_count, totalFalses;

    if (idx < N) {
	    // split based on least significant bit
	    quotient[idx] = values[idx];
	    for (int x = 0; x < loop + 1; ++x) {
		    remainder[idx] = quotient[idx] % 10;
		    quotient[idx] = quotient[idx] / 10;
	    }

	    // set e[idx] = 0 in each 1 input and e[idx] = 1 in each 0 input	
	    if (remainder[idx] == 1) {
		    split[idx] = 1;
		    e[idx] = 0;
	    }
	    else {
		    split[idx] = 0;
		    e[idx] = 1;
	    }
    }
    __syncthreads();

    if (idx < N) {
	    // scan the 1s
	    f_count = 0;
	    for (int x = 0; x < N; ++x) {
		    f[x] = f_count;
		    if (e[x] == 1)
			    f_count++;
	    }

	    // calculate totalFalses
	    totalFalses = e[N-1] + f[N-1];

	    if (split[idx] == 1) {
		    // t = idx - f + totalFalses
		    t[idx] = idx - f[idx] + totalFalses;
	    }
	    else if (split[idx] == 0) {
		    // t = f[idx]
		    t[idx] = f[idx];
	    }

	    // Scatter input using t as scatter address
	    temp[t[idx]] = values[idx];
    }
    __syncthreads();

    // copy new arrangement back to values
    if (idx < N) {
	    values[idx] = temp[idx];
    }
}

 // program main
 int main(int argc, char** argv) {
 
 if (argc != 2 ) {
	    fprintf(stderr, "Syntax: %s  <CacheConfL1> \n", argv[0]);
    		return EXIT_FAILURE;
    }

  	int CacheConfL1 = atoi(argv[1]);
	size_t size = N * sizeof(int);

    printf("./radix_sort starting with %d numbers...\n", N);
    //unsigned int hTimer;

    int devId = 0;
    checkCuda( hipSetDevice(devId) );
    hipDeviceReset();

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId) );
  	printf("Device: %s\n", prop.name);
  	
  	
    // allocate host memory
    r_values = (int*)malloc(size);

    // allocate device memory
     hipMalloc((void**)&d_values, size) ;
     hipMalloc((void**)&t_values, size) ;
     hipMalloc((void**)&d_split, size) ;
     hipMalloc((void**)&d_e, size) ;
     hipMalloc((void**)&d_f, size) ;
     hipMalloc((void**)&d_t, size) ;

    /* Types of data sets to be sorted:
     *	1. Normal distribution
     *	2. Gaussian distribution
     *	3. Bucket distribution
     * 	4. Sorted distribution
     *	5. Zero distribution
     */

    for (int i = 0; i < 5; ++i) {
	    // Initialize data set
	    Init(r_values, i);

	    // copy data to device
	     hipMemcpy(d_values, r_values, size, hipMemcpyHostToDevice) ;
	
	    printf("Beginning kernel execution...\n");
	
	    if (CacheConfL1 == 1){
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(Radix_sort), hipFuncCachePreferShared);
        }
        else if (CacheConfL1 == 2){
    	    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Radix_sort), hipFuncCachePreferEqual);
        }
        else if (CacheConfL1 == 3){
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(Radix_sort), hipFuncCachePreferL1);
        }
        else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(Radix_sort), hipFuncCachePreferNone);
        }
        
	    hipDeviceSynchronize() ;



	    // execute kernel
	    for (int j = 0; j < 8; ++j) {
            hipProfilerStart(); 
		    Radix_sort <<< 1, N >>> (d_values, t_values, j, d_split, d_e, d_f, d_t);
	    }
	    //cutilCheckMsg( "Kernel execution failed...\n" );

	     hipDeviceSynchronize() ;
	
	    //double gpuTime = cutGetTimerValue(hTimer);

	    //printf("\nKernel execution completed in %f ms\n", gpuTime);

	    // copy data back to host
	    hipMemcpy(r_values, t_values, size, hipMemcpyDeviceToHost) ;

	    // convert to decimal & test print
	    int r;
	    for (int x = 0; x < N; ++x) {
		    int 	val = 0,
			    power = 0;
		    while (r_values[x] > 0) {
			    r = r_values[x] % 10;
			    val = val + int(pow(2.0, power) * r);
			    r_values[x] = r_values[x] / 10;
			    power++;
		    }
		    r_values[x] = val;
		    //printf("%d ", r_values[x]);
	    }
	    //printf("\n");

	    // test
	    printf("\nTesting results...\n");
	    for (int x = 0; x < N - 1; x++) 
            assert(r_values[x] <= r_values[x + 1]);
    }

    // free memory
    hipFree(d_values);
    hipFree(t_values);
    hipFree(d_split) ;
    hipFree(d_e) ;
    hipFree(d_f) ;
    hipFree(d_t) ;
    free(r_values);

    //cutilExit(argc, argv);
    hipDeviceReset();
 }
