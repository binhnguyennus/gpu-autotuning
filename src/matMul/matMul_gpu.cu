
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width, int Tile_Width) {
  float Pvalue = 0.0;

  int j = blockIdx.x * Tile_Width + threadIdx.x;
  int i = blockIdx.y * Tile_Width + threadIdx.y;

  for (int k = 0; k < Width; ++k) {
    Pvalue += Md[i * Width + k] * Nd[k * Width + j];
  }

  Pd[i * Width + j] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{

  if (argc != 4) {
    fprintf(stderr, "Syntax: %s <matrix size Width> < Block_size>  <CacheConfL1> \n", argv[0]);
    return EXIT_FAILURE;
  }

  int Width = atoi(argv[1]);
  int BlockSize = atoi(argv[2]);
  int devId = 0;
  int CacheConfL1 = atoi(argv[3]);

  checkCuda( hipSetDevice(devId) );
  hipDeviceReset();

  // allocate host memory for matrices M and N
  printf("Allocate host memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));
  float Pt[Width*Width];

  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  checkCuda( hipMalloc((void**) &Md, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Nd, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Pd, Width * Width * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");
  if (CacheConfL1 == 1){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferShared);
  }
  else if (CacheConfL1 == 2){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferEqual);
  }
  else if (CacheConfL1 == 3){
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferL1);
  }
  else {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul), hipFuncCachePreferNone);
  }


  int GridSize = (Width + BlockSize-1) / BlockSize;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(BlockSize, BlockSize);

  hipProfilerStart();
  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width, BlockSize);
  hipProfilerStop();

  // copy result from device to host
  checkCuda( hipMemcpy( P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);


  //Assert Process
  char fileName[20] = "../matMul/matMul_";
  char bufferWidth[5] = " ";
  sprintf(bufferWidth, "%d", Width);
  strcat(fileName, bufferWidth);
  strcat(fileName, ".out");
  
  FILE *ptr_file;
  ptr_file =fopen(fileName, "r");
  if (!ptr_file) return 1;

  for (int i=0; i < Width; i++){
      for (int j=0; j < Width; j++){ 
	fscanf(ptr_file, "%f", &Pt[i * Width + j]);
      }
  }
  fclose(ptr_file); 

    for(int i=0 ;i<Width; i++) {
        for(int j=0; j<Width; j++) {
	   assert(fabs(P[i * Width + j] - Pt[i * Width + j]) < 0.01);
        }
    }




  // clean up memory
  free(M);
  free(N);
  free(P);
  checkCuda( hipFree(Md) );
  checkCuda( hipFree(Nd) );
  checkCuda( hipFree(Pd) );

  return 0;
}

